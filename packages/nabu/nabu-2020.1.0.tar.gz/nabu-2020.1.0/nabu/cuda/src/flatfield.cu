#include "hip/hip_runtime.h"
#ifndef N_FLATS
    #error "Please provide the N_FLATS variable"
#endif

#ifndef N_DARKS
    #error "Please provide the N_FLATS variable"
#endif



/**
 * In-place flat-field normalization.
 * This kernel assumes that all the radios are loaded into memory
 * (although not necessarily the full radios images)
 * and in radios[x, y z], z in the radio index
 *
 * radios: 3D array
 * flats: 3D array
 * darks: 3D array
 * Nx: number of pixel horizontally in the radios
 * Nx: number of pixel vertically in the radios
 * Nx: number of radios
 * flats_indices: indices of flats, in sorted order
 * darks_indices: indices of darks, in sorted order
 **/
__global__ void flatfield_normalization(
    float* radios,
    float* flats,
    float* darks,
    int Nx,
    int Ny,
    int Nz,
    int* flats_indices,
    int* darks_indices,
    int* radios_indices
) {
    uint x = blockDim.x * blockIdx.x + threadIdx.x;
    uint y = blockDim.y * blockIdx.y + threadIdx.y;
    uint z = blockDim.z * blockIdx.z + threadIdx.z;
    if ((x >= Nx) || (y >= Ny) || (z >= Nz)) return;
    uint pos = (z*Ny+y)*Nx + x;
    int radio_idx = radios_indices[z];

    float dark_val = 0.0f, flat_val = 1.0f;

    #if N_FLATS == 1
        flat_val = flats[y*Nx + x];
    #else
        // interpolation between 2 flats
        for (int i = 0; i < N_FLATS-1; i++) {
            int ind_prev = flats_indices[i];
            int ind_next = flats_indices[i+1];
            if (ind_prev >= radio_idx) {
                flat_val = flats[(i*Ny+y)*Nx + x];
                break;
            }
            else if (ind_prev < radio_idx && radio_idx < ind_next) {
                // Linear interpolation
                // TODO nearest interpolation
                int delta = ind_next - ind_prev;
                float w1 = 1.0f - (radio_idx*1.0f - ind_prev) / delta;
                float w2 = 1.0f - (ind_next*1.0f - radio_idx) / delta;
                flat_val = w1 * flats[(i*Ny+y)*Nx + x] + w2 * flats[((i+1)*Ny+y)*Nx + x];
                break;
            }
            else if (ind_next <= radio_idx) {
                flat_val = flats[((i+1)*Ny+y)*Nx + x];
                break;
            }
        }
    #endif
    #if (N_DARKS == 1)
        dark_val = darks[y*Nx + x];
    #else
        // TODO interpolate between darks
        // Same as above...
        #error "N_DARKS > 1 is not supported yet"
    #endif

    radios[pos] = (radios[pos] - dark_val) / (flat_val - dark_val);
}
